#include "hip/hip_runtime.h"
// Header libraries - CUDA part lives in "device_fft.h"
#include "audio_extractor.h"
#include "device_fft.h"
#include "piano_scale_draw.h"

#include <QApplication>
#include <QProcessEnvironment>
#include <QString>
#include <fstream>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <qwt_plot.h>
#include <qwt_plot_curve.h>
#include <qwt_plot_grid.h>
#include <qwt_plot_renderer.h>
#include <qwt_scale_draw.h>
#include <qwt_scale_engine.h>
#include <string>

#define WINDOW_SIZE (8192)
#define BATCH_SIZE (1024)
#define FREQUENCY_CUTOFF (4200) // In Hz, piano C8 = 4186.01

void write_csv(std::string filename, std::vector<float> &data)
{
    nvtx3::scoped_range r(("Write results: " + filename).c_str());
    std::ofstream outfile(filename);
    if (outfile.is_open()) {
        std::ostringstream buffer;
        for (const auto &val : data) {
            buffer << val << "\n";
        }
        outfile << buffer.str();
        outfile.close();
    }
    else {
        std::cout << "Unable to open file " << filename << std::endl;
    }
}

void draw_spectrogram_animation(const std::string &filename, const std::unique_ptr<DeviceFFT> &fft)
{
    int num_bins = fft->get_num_bins();

    std::vector<double> frequencies(num_bins);
    for (int i = 0; i < frequencies.size(); i++) {
        frequencies[i] = i * fft->get_hz_per_bin();
    }

    double max = fft->get_output_max();

    nvtxRangePushA("Set up plot");
    // Create the plot
    QwtPlot plot;
    plot.setCanvasBackground(Qt::white);
    plot.setTitle("Frequency");
    plot.setAxisTitle(QwtPlot::xBottom, "Frequency (Hz)");
    plot.setAxisTitle(QwtPlot::yLeft, "Magnitude");
    plot.setAxisScaleEngine(QwtPlot::xBottom, new QwtLogScaleEngine());

    QwtScaleDiv scale(27.0, 4200.0, QList<double>(), QList<double>(), notes);
    plot.setAxisScale(QwtPlot::xBottom, 27, 4200); // Frequency range
    plot.setAxisScaleDiv(QwtPlot::xBottom, scale);
    plot.setAxisScaleDraw(QwtPlot::xBottom, new PianoScaleDraw());
    plot.setAxisScale(QwtPlot::yLeft, 0, max);
    plot.setFixedSize(1920, 1080);

    // Initialize grid: dotted gray lines for major, attach to plot
    std::unique_ptr<QwtPlotGrid> grid = std::make_unique<QwtPlotGrid>();
    grid->setMajorPen(QPen(Qt::gray, 0, Qt::DashLine));
    grid->attach(&plot);

    std::unique_ptr<QwtPlotCurve> curve = std::make_unique<QwtPlotCurve>();
    curve->setPen(QPen(Qt::blue, 3, Qt::SolidLine));
    curve->attach(&plot);

    QwtPlotRenderer renderer;
    renderer.setDiscardFlag(QwtPlotRenderer::DiscardBackground, false);
    nvtxRangePop();

    nvtxRangePushA("Initialize video writer");
    cv::VideoWriter video(filename,
                          cv::VideoWriter::fourcc('a', 'v', 'c', '1'),
                          (double)fft->get_sample_rate() / (double)fft->get_hop_size(),
                          cv::Size(1920, 1080));
    if (!video.isOpened()) {
        std::cerr << "Could not open the video writer" << std::endl;
        return;
    }
    nvtxRangePop();

    QPixmap pixmap(plot.size());
    QPainter painter(&pixmap);
    QImage image = pixmap.toImage();
    cv::Mat mat(image.height(), image.width(), CV_8UC4);
    cv::Mat bgr_mat;

    for (int frame = 0; frame < fft->_host_output_buf.size() / num_bins; ++frame) {
        nvtx3::scoped_range r("Process frame " + std::to_string(frame));
        if (frame % 100 == 0) {
            std::cout << "Processing frame: " << frame << "/" << fft->_host_output_buf.size() / num_bins << std::endl;
        }

        // Set samples for the curve (this may involve allocations depending on curve implementation)
        curve->setSamples(frequencies.data(),
                          static_cast<double *>(thrust::raw_pointer_cast(fft->_host_output_buf.data())) + frame * num_bins,
                          num_bins);

        // Render plot without recreating pixmap, painter, or QImage
        plot.replot();
        renderer.render(&plot, &painter, plot.geometry());
        image = pixmap.toImage(); // Reset the image without reallocating pixmap

        {
            nvtx3::scoped_range r2("Write mat to video");
            mat = cv::Mat(image.height(), image.width(), CV_8UC4, const_cast<uchar *>(image.bits()), image.bytesPerLine());
            cv::cvtColor(mat, bgr_mat, cv::COLOR_BGRA2BGR); // Reuse bgr_mat
            video.write(bgr_mat);                           // Write the frame to the video
        }
    }
}

void draw_spectrogram_image(const std::string &filename, const std::unique_ptr<DeviceFFT> &fft, size_t full_length, size_t hop_size)
{
    nvtx3::scoped_range r("Image Creation");

    cv::Mat image(full_length / hop_size, fft->get_num_bins(), CV_32F, fft->_host_output_buf.data().get());
    cv::Mat normalized_image;
    cv::normalize(image, normalized_image, 0, 255, cv::NORM_MINMAX, CV_32F);

    cv::Mat grayscale;
    normalized_image.convertTo(grayscale, CV_8U);
    cv::Mat colorized;
    cv::applyColorMap(grayscale, colorized, cv::COLORMAP_JET);

    cv::imwrite(filename, colorized);
}

int main(int argc, char **argv)
{
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file>" << std::endl;
        return EXIT_FAILURE;
    }

    qputenv("QT_QPA_PLATFORM", QByteArray("offscreen"));
    QApplication app(argc, argv);

    // Initialize defice FFT
    std::unique_ptr<DeviceFFT> fft = nullptr;

    std::string video_file = argv[1];

    std::cout << "Processing data for file: " << video_file << std::endl;

    bool first         = true;
    size_t full_length = 0;
    size_t frame_count = 0;
    int hop_size       = 0;

    try {
        // Initialize the audio extractor and read frames. The decoder will throw an exception on error.
        AudioExtractor extractor(video_file);
        AVFrame *frame = nullptr;

        while (extractor.read_frame(frame)) {
            if (first) {
                // On first frame we'll learn the data sizes and formats, so we can initialize the FFT
                AVSampleFormat sample_fmt = static_cast<AVSampleFormat>(frame->format);
                std::cout << "Sample format: " << av_get_sample_fmt_name(sample_fmt);
                std::cout << " - " << (av_sample_fmt_is_planar(sample_fmt) ? "Planar" : "Non-Planar") << std::endl;

                if (!av_sample_fmt_is_planar(sample_fmt)) {
                    std::cerr << "ERROR: Only planar audio formats are supported" << std::endl;
                    return EXIT_FAILURE;
                }

                // Determine full memory requirement, dependent parameters, and initialize
                full_length = extractor.total_frames() * frame->nb_samples;
                std::cout << "Sample rate: " << frame->sample_rate << " kHz" << std::endl;
                std::cout << "Total: " << full_length << " samples across " << extractor.total_frames() << " frames" << std::endl;
                hop_size = frame->sample_rate / 60;
                fft      = std::make_unique<DeviceFFT>(WINDOW_SIZE, hop_size, BATCH_SIZE, full_length, frame->sample_rate, FREQUENCY_CUTOFF);
                first    = false;
            }

            size_t data_size = extractor.bytes_per_sample() * frame->nb_samples;
            frame_count++;
            bool last_frame = (frame_count == (extractor.total_frames() - 1)) ? true : false;
            if (fft->add_sample(reinterpret_cast<float *>(frame->data[0]), frame->nb_samples, last_frame)) {
                fft->run_batch_fft(true, true);
            }
        }
    }
    catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    // Wait for all threads to finish
    fft->synchronize();

    // Uncomment below to write FFT output values to CSV
    // write_csv("raw.csv", fft->_host_output_buf);

    // Uncomment below to write an animation of all FFT output values across frames
    draw_spectrogram_animation("animation.mp4", fft);

    // Uncomment below to save the entire spectrogram as a PNG image
    draw_spectrogram_image("spectrogram.png", fft, full_length, hop_size);

    return EXIT_SUCCESS;
}
