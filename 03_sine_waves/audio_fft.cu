#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hipfft/hipfft.h>
#include <fstream>
#include <iostream>
#include <nvtx3/nvToolsExt.h>
#include <opencv2/opencv.hpp>
#include <string>
#include <thrust/device_vector.h>

// Include FFMPeg headers
extern "C"
{
#include <libavcodec/avcodec.h>
#include <libavformat/avformat.h>
#include <libavutil/channel_layout.h>
#include <libavutil/samplefmt.h>
}

#define WINDOW_SIZE (4096)
#define HOP_SIZE (1024)

class DeviceFFT
{
private:
    thrust::device_vector<float> _fft_buf;
    thrust::device_vector<float> _hann;
    thrust::device_vector<hipfftComplex> _fft_results;
    thrust::device_vector<float> _mags;

    hipStream_t _data_xfer_stream = 0;
    hipStream_t _compute_stream   = 0;

    size_t _buf_depth     = 0;
    int _last_window_size = 0;


public:
    void
    normalize_values()
    {
        NVTX3_FUNC_RANGE();

        float *d_max              = nullptr;
        void *d_temp_storage      = nullptr;
        size_t temp_storage_bytes = 0;

        {
            nvtx3::scoped_range find_max("Find max");
            auto abs_it = thrust::transform_iterator(_fft_buf.begin(),
                                                     [] __host__ __device__(float x) {
                                                         return cuda::std::abs(x);
                                                     });

            hipMalloc(&d_max, sizeof(float));
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, abs_it, d_max, _fft_buf.size());
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, abs_it, d_max, _fft_buf.size());
            hipFree(d_temp_storage);
        }

        {
            nvtx3::scoped_range norm("Normalization");

            auto normalize_op = [d_max] __device__(float &x) {
                x = x / *d_max;
            };

            temp_storage_bytes = 0;
            d_temp_storage     = nullptr;
            hipcub::DeviceFor::ForEach(d_temp_storage, temp_storage_bytes, _fft_buf.begin(), _fft_buf.end(), normalize_op);
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            hipcub::DeviceFor::ForEach(d_temp_storage, temp_storage_bytes, _fft_buf.begin(), _fft_buf.end(), normalize_op);
            hipFree(d_temp_storage);

            hipFree(d_max);
        }
    }


    struct hann_functor
    {
        size_t window_size;

        __host__ __device__
        hann_functor(size_t _window_size) : window_size(_window_size) {}

        __host__ __device__ float operator()(const int &index) const
        {
            return 0.5f * (1.0f - cosf((2.0f * M_PI * index) / window_size));
        }
    };


    DeviceFFT(size_t buffer_size, size_t window_size) : _fft_buf(buffer_size),
                                                        _hann(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), hann_functor(window_size)),
                                                              thrust::make_transform_iterator(thrust::counting_iterator<int>(window_size), hann_functor(window_size)))
    {
        NVTX3_FUNC_RANGE();
        hipStreamCreateWithFlags(&_data_xfer_stream, hipStreamNonBlocking);
        hipStreamCreateWithFlags(&_compute_stream, hipStreamNonBlocking);
    }

    ~DeviceFFT()
    {
        NVTX3_FUNC_RANGE();
        hipStreamDestroy(_data_xfer_stream);
        hipStreamDestroy(_compute_stream);
    }

    void add_sample(float *data, int length)
    {
        if (_buf_depth + length <= _fft_buf.size()) {
            nvtx3::scoped_range copy_in("Copy in " + std::to_string(length) + " from " + std::to_string(_buf_depth).c_str());
            hipMemcpyAsync(thrust::raw_pointer_cast(_fft_buf.data()) + _buf_depth,
                            data,
                            sizeof(float) * length,
                            hipMemcpyHostToDevice,
                            _data_xfer_stream);
            _buf_depth += length;
        }
        else {
            nvtx3::scoped_range("Shuffle and copy in");

            hipMemcpyAsync(thrust::raw_pointer_cast(_fft_buf.data()),
                            thrust::raw_pointer_cast(_fft_buf.data() + length),
                            (_buf_depth - length) * sizeof(float),
                            hipMemcpyDeviceToDevice, _data_xfer_stream);
            hipMemcpyAsync(thrust::raw_pointer_cast(_fft_buf.data()) + _buf_depth - length,
                            data,
                            sizeof(float) * length,
                            hipMemcpyHostToDevice,
                            _data_xfer_stream);
        }
    }

    std::vector<float> get_window_vector()
    {
        std::vector<float> window(_hann.size());
        thrust::copy(_hann.begin(), _hann.end(), window.begin());

        return window;
    }

    hipfftResult run_batch_fft(int length, int window_size, int hop, bool normalize = true)
    {
        NVTX3_FUNC_RANGE();
        hipfftResult result;

        nvtxRangePushA("Ensure data transfer completion");
        hipStreamSynchronize(_data_xfer_stream);
        nvtxRangePop();


        int _num_batches  = (length - window_size) / hop + 1;
        _last_window_size = window_size;
        {
            nvtx3::scoped_range r("Allocate intermediate memory");
            _fft_results.resize(_num_batches * window_size);
            _mags.resize(_num_batches * window_size);
        }

        if (normalize) {
            normalize_values();
        }

        // Create cuFFT plan
        hipfftHandle plan;
        {
            nvtx3::scoped_range plan_creation("Create FFT Plan");
            int n[1]       = {window_size}; // FFT size is 4096
            int inembed[1] = {length};      // Input size per batch
            int istride    = 1;             // Stride between samples in a batch
            int idist      = hop;           // Distance between the start of each batch (hop size)
            int onembed[1] = {window_size}; // Output size per batch (same as input in 1D FFT)
            int ostride    = 1;
            int odist      = window_size; // Distance between the start of each output batch

            // Batch mode FFT plan creation
            hipfftPlanMany(&plan, 1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, _num_batches);
            hipfftSetStream(plan, _compute_stream);
        }

        {
            nvtx3::scoped_range rng_exec("Execute");
            result = hipfftExecR2C(plan, _fft_buf.data().get(), _fft_results.data().get());
            if (result != HIPFFT_SUCCESS) {
                std::cout << "CUFFT Error: ExecR2C failed" << std::endl;
                return result;
            }
        }

        return result;
    }

    std::vector<float> get_magnitudes()
    {
        NVTX3_FUNC_RANGE();
        hipStreamSynchronize(_compute_stream);

        {
            nvtx3::scoped_range r("Convert bins to magnitude");
            auto mag = thrust::transform_iterator(_fft_results.begin(),
                                                  [this] __host__ __device__(hipfftComplex c) {
                                                      return sqrtf(c.x * c.x + c.y * c.y) / _last_window_size;
                                                  });

            thrust::copy(mag, mag + _fft_results.size(), _mags.begin());
        }

        nvtxRangePushA("Allocate host vector");
        std::vector<float> h_mags(_mags.size());
        nvtxRangePop();

        nvtxRangePushA("Copy to Host");
        thrust::copy(_mags.begin(), _mags.end(), h_mags.begin());
        nvtxRangePop();

        return h_mags;
    }

    std::vector<std::uint8_t> get_img_magnitudes()
    {
        NVTX3_FUNC_RANGE();
        hipStreamSynchronize(_compute_stream);
        nvtxRangePushA("Transform FFT results");
        {
            nvtx3::scoped_range r("Transform FFT results");

            auto mag = thrust::transform_iterator(_fft_results.begin(),
                                                  [this] __host__ __device__(hipfftComplex c) {
                                                      return sqrtf(c.x * c.x + c.y * c.y) / _last_window_size;
                                                  });

            thrust::copy(mag, mag + _fft_results.size(), _mags.begin());
        }

        float *d_max              = nullptr;
        void *d_temp_storage      = nullptr;
        size_t temp_storage_bytes = 0;
        {
            nvtx3::scoped_range r("Find Maximum");

            hipMalloc(&d_max, sizeof(float));
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, _mags.data().get(), d_max, _mags.size());
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, _mags.data().get(), d_max, _mags.size());
            hipFree(d_temp_storage);
        }

        nvtxRangePushA("Convert to normalized uchar");
        const int k        = 25;
        auto img_transform = thrust::transform_iterator(_mags.begin(),
                                                        [d_max, k] __host__ __device__(float x) {
                                                            return (std::uint8_t)((logf(1 + (k * (x / *d_max))) / logf(1 + k)) * 255);
                                                        });
        thrust::device_vector<std::uint8_t> img_mags(img_transform, img_transform + _mags.size());
        nvtxRangePop();

        nvtxRangePushA("Allocate host memory");
        std::vector<std::uint8_t> img(img_mags.size());
        nvtxRangePop();

        nvtxRangePushA("Copy to Host");
        thrust::copy(img_mags.begin(), img_mags.end(), img.begin());
        nvtxRangePop();

        hipFree(d_max);

        return img;
    }
};

void write_csv(std::string filename, std::vector<float> &data)
{
    nvtx3::scoped_range r(("Write results: " + filename).c_str());
    std::ofstream outfile(filename);
    if (outfile.is_open()) {
        std::ostringstream buffer;
        for (const auto &val : data) {
            buffer << val << "\n";
        }
        outfile << buffer.str();
        outfile.close();
    }
    else {
        std::cout << "Unable to open file " << filename << std::endl;
    }
}

int main(int argc, char **argv)
{
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file>" << std::endl;
        return EXIT_FAILURE;
    }

    // Initialize defice FFT
    std::unique_ptr<DeviceFFT> fft = nullptr;

    std::string video_file = argv[1];

    std::cout << "Processing data for file: " << video_file << std::endl;

    // Initialize FFmpeg libraries
    AVFormatContext *format_context = nullptr;

    // Open the video
    if (avformat_open_input(&format_context, video_file.c_str(), nullptr, nullptr) < 0) {
        std::cerr << "Error opening input file" << std::endl;
        return EXIT_FAILURE;
    }

    // Retrieve stream information
    if (avformat_find_stream_info(format_context, nullptr) < 0) {
        std::cerr << "Error finding stream info" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    const AVCodec *codec          = nullptr;
    AVCodecContext *codec_context = nullptr;
    int audio_stream_index        = -1;

    // Find the first audio stream
    for (int i = 0; i < format_context->nb_streams; i++) {
        if (format_context->streams[i]->codecpar->codec_type == AVMEDIA_TYPE_AUDIO) {
            audio_stream_index = i;
            break;
        }
    }

    // Check if an audio stream was found
    if (audio_stream_index < 0) {
        std::cerr << "No audio stream found" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    AVCodecParameters *codec_parameters = format_context->streams[audio_stream_index]->codecpar;
    // Find the decoder for the audio stream
    codec = avcodec_find_decoder(codec_parameters->codec_id);
    if (!codec) {
        std::cerr << "Error finding decoder" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    // Allocate a codec context for the decoder
    codec_context = avcodec_alloc_context3(codec);
    if (!codec_context) {
        std::cerr << "Error allocating codec context" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    // Set the codec parameters to the allocated codec context
    if (avcodec_parameters_to_context(codec_context, codec_parameters) < 0) {
        std::cerr << "Error setting codec parameters" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    // Open the decoder
    if (avcodec_open2(codec_context, codec, nullptr) < 0) {
        std::cerr << "Error opening decoder" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    AVPacket *packet = av_packet_alloc();
    if (!packet) {
        std::cerr << "Error allocating packet" << std::endl;
        avcodec_free_context(&codec_context);
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }
    AVFrame *frame = av_frame_alloc();
    if (!frame) {
        std::cerr << "Error allocating frame" << std::endl;
        av_packet_free(&packet);
        avcodec_free_context(&codec_context);
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }


    bool first         = true;
    size_t full_length = 0;


    while (av_read_frame(format_context, packet) >= 0) {
        if (packet->stream_index == audio_stream_index) {
            int response = avcodec_send_packet(codec_context, packet);
            if (response < 0) {
                std::cerr << "Error sending packet to decoder" << std::endl;
                break;
            }

            while (response >= 0) {
                response = avcodec_receive_frame(codec_context, frame);
                if (response == AVERROR(EAGAIN) || response == AVERROR_EOF) {
                    break;
                }
                else if (response < 0) {
                    std::cerr << "Error during decoding" << std::endl;
                    av_packet_unref(packet);
                    return EXIT_FAILURE;
                }

                if (first) {
                    AVSampleFormat sample_fmt = static_cast<AVSampleFormat>(frame->format);
                    std::cout << "Sample format: " << av_get_sample_fmt_name(sample_fmt);
                    std::cout << " - " << (av_sample_fmt_is_planar(sample_fmt) ? "Planar" : "Non-Planar") << std::endl;

                    // Determine full memory requirement
                    full_length = format_context->streams[audio_stream_index]->nb_frames *
                                  frame->nb_samples;
                    std::cout << "Total: " << full_length << " samples across " << format_context->streams[audio_stream_index]->nb_frames << " frames" << std::endl;
                    fft   = std::make_unique<DeviceFFT>(full_length, WINDOW_SIZE);
                    first = false;
                }

                size_t data_size = av_get_bytes_per_sample(codec_context->sample_fmt) * frame->nb_samples;

                fft->add_sample(reinterpret_cast<float *>(frame->data[0]), frame->nb_samples);
            }
        }
        av_packet_unref(packet);
    }

    avformat_close_input(&format_context);


    // Now all data resides on the GPU. We can calculate the FFT
    hipfftResult result;

    // Save the Hann window for visualization later

    result = fft->run_batch_fft(full_length, WINDOW_SIZE, HOP_SIZE);
    if (result != HIPFFT_SUCCESS) {
        return EXIT_FAILURE;
    }


    std::vector<float> float_mags = fft->get_magnitudes();
    write_csv("video_mags.csv", float_mags);

    std::vector<std::uint8_t> mags = fft->get_img_magnitudes();

    std::vector<float> window = fft->get_window_vector();
    write_csv("window.csv", window);

    /*
        nvtxRangePushA("Image creation");

        cv::Mat image(full_length / HOP_SIZE, WINDOW_SIZE, CV_8UC1, mags.data());
        nvtxRangePushA("Crop");
        cv::Mat cropped_image = image(cv::Rect(0, 0, 400, image.rows));
        nvtxRangePop();
        nvtxRangePushA("Transpose");
        // cv::rotate(cropped_image, cropped_image, cv::ROTATE_90_COUNTERCLOCKWISE);
        cv::Mat color_image;

        // Apply a colormap. Options include COLORMAP_JET, COLORMAP_HOT, COLORMAP_COOL, etc.
        // cv::applyColorMap(cropped_image, color_image, cv::COLORMAP_JET);
        nvtxRangePop();

        nvtxRangePushA("Write");

        cv::imwrite("fft.png", cropped_image);
        nvtxRangePop();
        nvtxRangePop();
    */
    return EXIT_SUCCESS;
}
