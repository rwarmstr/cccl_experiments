#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hipfft/hipfft.h>
#include <fstream>
#include <iostream>
#include <nvtx3/nvToolsExt.h>
#include <opencv2/opencv.hpp>
#include <string>
#include <thrust/device_vector.h>

// Include FFMPeg headers
extern "C"
{
#include <libavcodec/avcodec.h>
#include <libavformat/avformat.h>
#include <libavutil/channel_layout.h>
#include <libavutil/samplefmt.h>
}

#define WINDOW_SIZE (4096)
#define HOP_SIZE (1024)

class DeviceFFT
{
private:
    thrust::device_vector<float> _fft_buf;
    thrust::device_vector<float> _hann;
    thrust::device_vector<hipfftComplex> _fft_results;

    size_t _buf_depth     = 0;
    int _last_window_size = 0;

public:
    void normalize_values()
    {
        nvtxRangePushA("Normalize input values");
        float *d_max              = nullptr;
        void *d_temp_storage      = nullptr;
        size_t temp_storage_bytes = 0;

        nvtxRangePushA("Find max");
        auto abs_it = thrust::transform_iterator(_fft_buf.begin(),
                                                 [] __host__ __device__(float x) {
                                                     return cuda::std::abs(x);
                                                 });

        hipMalloc(&d_max, sizeof(float));
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, abs_it, d_max, _fft_buf.size());
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, abs_it, d_max, _fft_buf.size());
        hipFree(d_temp_storage);
        nvtxRangePop();

        float max_value;
        hipMemcpy(&max_value, d_max, sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        std::cout << "Max value in array: " << max_value << std::endl;

        nvtxRangePushA("Normalize");

        auto normalize_op = [d_max] __device__(float &x) {
            x = x / *d_max;
        };

        temp_storage_bytes = 0;
        d_temp_storage     = nullptr;
        hipcub::DeviceFor::ForEach(d_temp_storage, temp_storage_bytes, _fft_buf.begin(), _fft_buf.end(), normalize_op);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceFor::ForEach(d_temp_storage, temp_storage_bytes, _fft_buf.begin(), _fft_buf.end(), normalize_op);
        hipFree(d_temp_storage);

        hipFree(d_max);
        nvtxRangePop();
        nvtxRangePop();
    }


    struct hann_functor
    {
        size_t window_size;

        __host__ __device__
        hann_functor(size_t _window_size) : window_size(_window_size) {}

        __host__ __device__ float operator()(const int &index) const
        {
            return 0.5f * (1.0f - cosf((2.0f * M_PI * index) / window_size));
        }
    };


    DeviceFFT(size_t buffer_size, size_t window_size) : _fft_buf(buffer_size),
                                                        _hann(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), hann_functor(window_size)),
                                                              thrust::make_transform_iterator(thrust::counting_iterator<int>(window_size), hann_functor(window_size)))
    {
        nvtxRangePushA("Context creation");
        hipFree(0);
        nvtxRangePop();
    }

    void add_sample(float *data, int length)
    {
        if (_buf_depth + length <= _fft_buf.size()) {
            nvtxRangePushA(("Copy in " + std::to_string(length) + " from " + std::to_string(_buf_depth)).c_str());
            thrust::copy(data, data + length, _fft_buf.begin() + _buf_depth);
            _buf_depth += length;
            nvtxRangePop();
        }
        else {
            nvtxRangePushA("Shuffle");
            nvtxRangePushA(("Move " + std::to_string(length) + " to " + std::to_string(_buf_depth) + " to 0").c_str());

            hipMemcpyAsync(thrust::raw_pointer_cast(_fft_buf.data()),
                            thrust::raw_pointer_cast(_fft_buf.data() + length),
                            (_buf_depth - length) * sizeof(float),
                            hipMemcpyDeviceToDevice);
            nvtxRangePop();
            nvtxRangePushA("Copy new");
            thrust::copy(data, data + length, _fft_buf.begin() + _buf_depth - length);
            nvtxRangePop();
            nvtxRangePop();
        }
    }

    std::vector<float> get_window_vector()
    {
        std::vector<float> window(_hann.size());
        thrust::copy(_hann.begin(), _hann.end(), window.begin());

        return window;
    }

    hipfftResult run_batch_fft(int length, int window_size, int hop, bool normalize = true)
    {
        hipfftResult result;

        nvtxRangePushA("FFT");
        nvtxRangePushA("Allocate intermediate memory");
        int _num_batches  = (length - window_size) / hop + 1;
        _last_window_size = window_size;
        _fft_results.resize(_num_batches * window_size);
        // thrust::device_vector<hipfftComplex> fft_results(num_batches * window_size);
        nvtxRangePop();

        if (normalize) {
            normalize_values();
        }

        nvtxRangePushA("Create Plan");
        // Create cuFFT plan
        hipfftHandle plan;
        int n[1]       = {window_size}; // FFT size is 4096
        int inembed[1] = {length};      // Input size per batch
        int istride    = 1;             // Stride between samples in a batch
        int idist      = hop;           // Distance between the start of each batch (hop size)
        int onembed[1] = {window_size}; // Output size per batch (same as input in 1D FFT)
        int ostride    = 1;
        int odist      = window_size; // Distance between the start of each output batch

        // Batch mode FFT plan creation
        hipfftPlanMany(&plan, 1, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, _num_batches);
        nvtxRangePop();

        nvtxRangePushA("Execute");
        result = hipfftExecR2C(plan, _fft_buf.data().get(), _fft_results.data().get());
        if (result != HIPFFT_SUCCESS) {
            std::cout << "CUFFT Error: ExecR2C failed" << std::endl;
            return result;
        }
        nvtxRangePop();
        nvtxRangePop();

        return result;
    }

    std::vector<float> get_magnitudes()
    {
        nvtxRangePushA("Retrieve bin magnitudes");
        nvtxRangePushA("Transform FFT results");
        auto mag = thrust::transform_iterator(_fft_results.begin(),
                                              [this] __host__ __device__(hipfftComplex c) {
                                                  return sqrtf(c.x * c.x + c.y * c.y) / _last_window_size;
                                              });

        thrust::device_vector<float> mags(mag, mag + _fft_results.size());
        nvtxRangePop();

        nvtxRangePushA("Allocate host vector");
        std::vector<float> h_mags(mags.size());
        nvtxRangePop();

        nvtxRangePushA("Copy to Host");
        thrust::copy(mags.begin(), mags.end(), h_mags.begin());
        nvtxRangePop();
        nvtxRangePop();

        return h_mags;
    }

    std::vector<std::uint8_t> get_img_magnitudes()
    {
        nvtxRangePushA("Retrieve bin magnitudes");
        nvtxRangePushA("Transform FFT results");
        auto mag = thrust::transform_iterator(_fft_results.begin(),
                                              [this] __host__ __device__(hipfftComplex c) {
                                                  return sqrtf(c.x * c.x + c.y * c.y) / _last_window_size;
                                              });

        thrust::device_vector<float> mags(mag, mag + _fft_results.size());
        nvtxRangePop();

        nvtxRangePushA("Find Maximum");
        float *d_max              = nullptr;
        void *d_temp_storage      = nullptr;
        size_t temp_storage_bytes = 0;

        hipMalloc(&d_max, sizeof(float));
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, mags.data().get(), d_max, mags.size());
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, mags.data().get(), d_max, mags.size());
        hipFree(d_temp_storage);
        nvtxRangePop();

        nvtxRangePushA("Convert to normalized uchar");
        const int k        = 25;
        auto img_transform = thrust::transform_iterator(mags.begin(),
                                                        [d_max, k] __host__ __device__(float x) {
                                                            return (std::uint8_t)((logf(1 + (k * (x / *d_max))) / logf(1 + k)) * 255);
                                                        });
        thrust::device_vector<std::uint8_t> img_mags(img_transform, img_transform + mags.size());
        nvtxRangePop();

        nvtxRangePushA("Allocate host memory");
        std::vector<std::uint8_t> img(img_mags.size());
        nvtxRangePop();

        nvtxRangePushA("Copy to Host");
        thrust::copy(img_mags.begin(), img_mags.end(), img.begin());
        nvtxRangePop();
        nvtxRangePop();

        hipFree(d_max);

        return img;
    }
};

void write_csv(std::string filename, std::vector<float> &data)
{
    nvtxRangePushA(("Write results: " + filename).c_str());
    std::ofstream outfile(filename);
    if (outfile.is_open()) {
        for (auto &val : data) {
            outfile << val << "\n";
        }
        outfile.close();
    }
    else {
        std::cout << "Unable to open file " << filename << std::endl;
    }
    nvtxRangePop();
}

int main(int argc, char **argv)
{
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file>" << std::endl;
        return EXIT_FAILURE;
    }

    // Initialize defice FFT
    std::unique_ptr<DeviceFFT> fft = nullptr;

    std::string video_file = argv[1];

    std::cout << "Processing data for file: " << video_file << std::endl;

    // Initialize FFmpeg libraries
    AVFormatContext *format_context = nullptr;

    // Open the video
    if (avformat_open_input(&format_context, video_file.c_str(), nullptr, nullptr) < 0) {
        std::cerr << "Error opening input file" << std::endl;
        return EXIT_FAILURE;
    }

    // Retrieve stream information
    if (avformat_find_stream_info(format_context, nullptr) < 0) {
        std::cerr << "Error finding stream info" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    const AVCodec *codec          = nullptr;
    AVCodecContext *codec_context = nullptr;
    int audio_stream_index        = -1;

    // Find the first audio stream
    for (int i = 0; i < format_context->nb_streams; i++) {
        if (format_context->streams[i]->codecpar->codec_type == AVMEDIA_TYPE_AUDIO) {
            audio_stream_index = i;
            break;
        }
    }

    // Check if an audio stream was found
    if (audio_stream_index < 0) {
        std::cerr << "No audio stream found" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    AVCodecParameters *codec_parameters = format_context->streams[audio_stream_index]->codecpar;
    // Find the decoder for the audio stream
    codec = avcodec_find_decoder(codec_parameters->codec_id);
    if (!codec) {
        std::cerr << "Error finding decoder" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    // Allocate a codec context for the decoder
    codec_context = avcodec_alloc_context3(codec);
    if (!codec_context) {
        std::cerr << "Error allocating codec context" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    // Set the codec parameters to the allocated codec context
    if (avcodec_parameters_to_context(codec_context, codec_parameters) < 0) {
        std::cerr << "Error setting codec parameters" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    // Open the decoder
    if (avcodec_open2(codec_context, codec, nullptr) < 0) {
        std::cerr << "Error opening decoder" << std::endl;
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }

    AVPacket *packet = av_packet_alloc();
    if (!packet) {
        std::cerr << "Error allocating packet" << std::endl;
        avcodec_free_context(&codec_context);
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }
    AVFrame *frame = av_frame_alloc();
    if (!frame) {
        std::cerr << "Error allocating frame" << std::endl;
        av_packet_free(&packet);
        avcodec_free_context(&codec_context);
        avformat_close_input(&format_context);
        return EXIT_FAILURE;
    }


    bool first         = true;
    size_t full_length = 0;


    while (av_read_frame(format_context, packet) >= 0) {
        if (packet->stream_index == audio_stream_index) {
            int response = avcodec_send_packet(codec_context, packet);
            if (response < 0) {
                std::cerr << "Error sending packet to decoder" << std::endl;
                break;
            }

            while (response >= 0) {
                response = avcodec_receive_frame(codec_context, frame);
                if (response == AVERROR(EAGAIN) || response == AVERROR_EOF) {
                    break;
                }
                else if (response < 0) {
                    std::cerr << "Error during decoding" << std::endl;
                    av_packet_unref(packet);
                    return EXIT_FAILURE;
                }

                if (first) {
                    AVSampleFormat sample_fmt = static_cast<AVSampleFormat>(frame->format);
                    std::cout << "Sample format: " << av_get_sample_fmt_name(sample_fmt);
                    std::cout << " - " << (av_sample_fmt_is_planar(sample_fmt) ? "Planar" : "Non-Planar") << std::endl;

                    // Determine full memory requirement
                    full_length = format_context->streams[audio_stream_index]->nb_frames *
                                  frame->nb_samples;
                    std::cout << "Total: " << full_length << " samples across " << format_context->streams[audio_stream_index]->nb_frames << " frames" << std::endl;
                    fft   = std::make_unique<DeviceFFT>(full_length, WINDOW_SIZE);
                    first = false;
                }

                size_t data_size = av_get_bytes_per_sample(codec_context->sample_fmt) * frame->nb_samples;

                fft->add_sample(reinterpret_cast<float *>(frame->data[0]), frame->nb_samples);
            }
        }
        av_packet_unref(packet);
    }

    avformat_close_input(&format_context);


    // Now all data resides on the GPU. We can calculate the FFT
    hipfftResult result;

    // Save the Hann window for visualization later

    result = fft->run_batch_fft(full_length, WINDOW_SIZE, HOP_SIZE);
    if (result != HIPFFT_SUCCESS) {
        return EXIT_FAILURE;
    }


    std::vector<float> float_mags = fft->get_magnitudes();
    write_csv("video_mags.csv", float_mags);

    std::vector<std::uint8_t> mags = fft->get_img_magnitudes();

    std::vector<float> window = fft->get_window_vector();
    write_csv("window.csv", window);

    nvtxRangePushA("Image creation");

    cv::Mat image(full_length / HOP_SIZE, WINDOW_SIZE, CV_8UC1, mags.data());
    nvtxRangePushA("Crop");
    cv::Mat cropped_image = image(cv::Rect(0, 0, 400, image.rows));
    nvtxRangePop();
    nvtxRangePushA("Transpose");
    cv::rotate(cropped_image, cropped_image, cv::ROTATE_90_COUNTERCLOCKWISE);
    cv::Mat color_image;

    // Apply a colormap. Options include COLORMAP_JET, COLORMAP_HOT, COLORMAP_COOL, etc.
    cv::applyColorMap(cropped_image, color_image, cv::COLORMAP_JET);
    nvtxRangePop();

    nvtxRangePushA("Write");

    cv::imwrite("fft.png", color_image);
    nvtxRangePop();
    nvtxRangePop();

    return EXIT_SUCCESS;
}
