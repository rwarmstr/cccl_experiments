#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/tabulate.h>
#include <hipfft/hipfft.h>
#include <array>
#include <nvtx3/nvToolsExt.h>

#define NUM_SAMPLES (1024 * 1024)

#define FREQ_A (440.0f)
#define FREQ_E (659.255f)

#define SAMPLE_RATE (44100.0f)

struct sine_wave_functor
{
    float amplitude;
    float frequency;
    float phase;

    __host__ __device__
    sine_wave_functor(float _amplitude, float _frequency, float _phase)
        : amplitude(_amplitude), frequency(_frequency), phase(_phase) {}

    __host__ __device__ float operator()(const int &index) const
    {
        return amplitude * sinf(frequency * index + phase);
    }
};

struct add_waves
{
    __host__ __device__ float operator()(const thrust::tuple<float, float> &t) const
    {
        return thrust::get<0>(t) + thrust::get<1>(t);
    }
};

void write_results(std::string filename, std::vector<float> &data)
{
    nvtxRangePushA(("Write results: " + filename).c_str());
    std::ofstream outfile("filename");
    if (outfile.is_open())
    {
        for (auto &val : data)
        {
            outfile << val << "\n";
        }
        outfile.close();
    }
    else
    {
        std::cout << "Unable to open file " << filename << std::endl;
    }
    nvtxRangePop();
}

int main(void)
{
    // Quick call to hipFree to ensure context creation
    nvtxRangePushA("Context creation");
    hipFree(0);
    nvtxRangePop();

    // Create a device vector to hold out input waveform
    nvtxRangePushA("Memory Initialization");
    auto wave1 = thrust::make_transform_iterator(thrust::counting_iterator<int>(0), sine_wave_functor(1, 2 * M_PI * FREQ_A / SAMPLE_RATE, 0));
    auto wave2 = thrust::make_transform_iterator(thrust::counting_iterator<int>(0), sine_wave_functor(0.5, 2 * M_PI * FREQ_E / SAMPLE_RATE, 0));
    const auto waves = thrust::make_zip_iterator(thrust::make_tuple(wave1, wave2));
    const auto initializer = thrust::make_transform_iterator(waves, add_waves());
    thrust::device_vector<float> d_combined(initializer, initializer + NUM_SAMPLES);

    nvtxRangePop();

    // Now let's do an FFT on the combined waveform
    hipfftHandle plan;
    hipfftResult result;

    int complex_size = NUM_SAMPLES / 2 + 1;
    thrust::device_vector<hipfftComplex> d_fft(complex_size);

    nvtxRangePushA("FFT");
    nvtxRangePushA("Create Plan");
    result = hipfftPlan1d(&plan, NUM_SAMPLES, HIPFFT_R2C, 1);
    if (result != HIPFFT_SUCCESS)
    {
        std::cout << "CUFFT Error: Plan creation failed" << std::endl;
        return EXIT_FAILURE;
    }
    nvtxRangePop();
    nvtxRangePushA("Execute");

    result = hipfftExecR2C(plan, d_combined.data().get(), d_fft.data().get());
    if (result != HIPFFT_SUCCESS)
    {
        std::cout << "CUFFT Error: ExecR2C failed" << std::endl;
        return EXIT_FAILURE;
    }
    nvtxRangePop();
    nvtxRangePop();

    // On-device magnitude spectrum
    nvtxRangePushA("FFT bin magnitudes");
    auto mag = thrust::transform_iterator(d_fft.begin(), [] __host__ __device__(hipfftComplex c)
                                          { return sqrtf(c.x * c.x + c.y * c.y) / NUM_SAMPLES; });
    thrust::device_vector<float> d_mags{mag, mag + d_fft.size()};
    nvtxRangePop();

    nvtxRangePushA("Copy results");
    std::vector<float> h_combined(d_combined.size());
    std::vector<float> h_mags(d_mags.size());

    thrust::copy(d_combined.begin(), d_combined.end(), h_combined.begin());
    thrust::copy(d_mags.begin(), d_mags.end(), h_mags.begin());
    nvtxRangePop();

    // Write the results to a file
    write_results("combined.csv", h_combined);
    write_results("mags.csv", h_mags);

    return EXIT_SUCCESS;
}
